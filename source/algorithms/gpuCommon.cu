#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "gpuCommon.h"

__global__ void nopKernel(){
    //just nop-kernel
}

void nop_run(){
    nopKernel<<<1, 1>>>();
    CUDA_CHECK_ERROR;
}
