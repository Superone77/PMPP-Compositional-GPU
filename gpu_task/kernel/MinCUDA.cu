#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include "MinCUDA.h"

#define N 16
#define BLOCKSIZE 16

__global__ void minKernel(double *min, const double *a){
    __shared__ double mintile[BLOCKSIZE];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    mintile[tid] = a[i];
    __syncthreads();

    // strided index and non-divergent branch
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) {
            if (mintile[tid + s] < mintile[tid])
                mintile[tid] = mintile[tid + s];
        }
        __syncthreads();
}

    if (tid == 0) {
        min[blockIdx.x] = mintile[0];
    }
}
__global__ void finalminKernel(double *min) {
    __shared__ double mintile[BLOCKSIZE];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    mintile[tid] = min[i];
    __syncthreads();

    // strided index and non-divergent branch
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) {
            if (mintile[tid + s] < mintile[tid])
                mintile[tid] = mintile[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        min[blockIdx.x] = mintile[0];
    }
}

hipError_t MinCUDA::Compute(double*min, const double *a, float &time){
    double *dev_a = 0;
    double *dev_min = 0;
    float milliseconds = 0;
    dim3 dimBlock(BLOCKSIZE);
    dim3 dimGrid(N);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipSetDevice(0);
    CUDA_CHECK_ERROR;

    hipMalloc((void**)&dev_min, N * sizeof(float));
    CUDA_CHECK_ERROR;

    hipMalloc((void**)&dev_a, N * N * sizeof(float));
    CUDA_CHECK_ERROR;

    hipMemcpy(dev_a, a, N * N * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR;

    hipEventRecord(start);
    minKernel<<<dimGrid, dimBlock>>>(dev_min, dev_a);
    CUDA_CHECK_ERROR;
    hipDeviceSynchronize();
    finalminKernel<<<1, dimBlock>>>(dev_min);
    CUDA_CHECK_ERROR;
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //hipDeviceSynchronize();
    CUDA_CHECK_ERROR;
    hipMemcpy(min, dev_min, sizeof(double), hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR;
    hipEventElapsedTime(&milliseconds, start, stop);
    CUDA_CHECK_ERROR;
    time = milliseconds;


    return hipSuccess;
}



