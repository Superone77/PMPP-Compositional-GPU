#include "hip/hip_runtime.h"
/*
 * nop-kernel
 */
#include <stdio.h>
#include "NopCUDA.h"
#define BLOCK_SIZE 32


__global__ void nopKernel(){
    //just nop-kernel
}

void NopCUDA::Compute(float& time){
    float milliseconds = 0;
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(1,1);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    nopKernel<<<dimGrid, dimBlock>>>();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    time = milliseconds;
    //printf("nop-kernel\n");
    CUDA_CHECK_ERROR;
}